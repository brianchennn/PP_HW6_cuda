#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "kernel.h"

__global__ void convolution(
    int *filterWidth,
    float *outputImage,
    const float *filter,
    const float *inputImage
) 
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int x_size = gridDim.x;
    const int y_size = gridDim.y;
    int halffilterSize = *filterWidth / 2;
    float sum = 0.0;
    int k, l;
    for (k = -halffilterSize; k <= halffilterSize; k++)
    {
        for (l = -halffilterSize; l <= halffilterSize; l++)
        {
            if (iy + k >= 0 && iy + k < y_size &&
                    ix + l >= 0 && ix + l < x_size)
            {
                sum += inputImage[(iy + k) * x_size + ix + l] *
                    filter[(k + halffilterSize) * *filterWidth +
                    l + halffilterSize];
            }
        }
    }
    outputImage[iy * x_size + ix] = sum;    
}

void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
        float *inputImage, float *outputImage)
{
    int filterSize = filterWidth * filterWidth;

    // Create memory buffers on the device for each vector
    int *dev_filter_width;
    float *dev_filter, *dev_inputImage, *dev_outputImage;
    hipMalloc(&dev_filter_width, sizeof(int));
    hipMalloc(&dev_filter, filterSize * sizeof(float));
    hipMalloc(&dev_inputImage, imageHeight * imageWidth * sizeof(float));
    hipMalloc(&dev_outputImage, imageHeight * imageWidth * sizeof(float));
    
    // Copy the filter and inputImage to their respective memory buffers
    hipMemcpy(dev_filter_width, &filterWidth, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_filter, filter, filterSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_inputImage, inputImage, imageHeight * imageWidth * sizeof(float), hipMemcpyHostToDevice);

   
    // Execute the OpenCL kernel on the list
    dim3 threadPerBlock(10,10);
    dim3 numBlocks(imageWidth / threadPerBlock.x, imageHeight / threadPerBlock.y);
    convolution<<<numBlocks, threadPerBlock>>>(dev_filter_width, dev_outputImage, dev_filter, dev_inputImage);
    hipMemcpy(outputImage, dev_outputImage, imageHeight * imageWidth * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_filter_width);
    hipFree(dev_filter);
    hipFree(dev_inputImage);
    hipFree(dev_outputImage);
}
